#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2013 Opposite Renderer
 * For the full copyright and license information, please view the LICENSE.txt
 * file that was distributed with this source code.
*/
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "config.h"
#include "renderer/helpers/random.h"
#include "renderer/Light.h"
#include "renderer/RayType.h"
#include "renderer/Hitpoint.h"
#include "renderer/ShadowPRD.h"
#include "renderer/helpers/light.h"

using namespace optix;

rtDeclareVariable(rtObject, sceneRootObject, , );
rtBuffer<Hitpoint, 2> raytracePassOutputBuffer;
rtBuffer<float3, 2> directRadianceBuffer;
rtBuffer<RandomState, 2> randomStates;
rtBuffer<Light, 1> lights;
rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, );
rtDeclareVariable(ShadowPRD, shadowPrd, rtPayload, );

RT_PROGRAM void kernel()
{
    Hitpoint rec = raytracePassOutputBuffer[launchIndex];
    
    // Use radiance value if we do not hit a non-specular surface
    if(!(rec.flags & PRD_HIT_NON_SPECULAR))
    {
        if((rec.flags & PRD_HIT_EMITTER) && !(rec.flags & PRD_HIT_SPECULAR))
        {
            directRadianceBuffer[launchIndex] = fminf(rec.radiance, make_float3(1));
        }
        else
        {
            directRadianceBuffer[launchIndex] = rec.radiance;
        }

        return;
    }

    /*
    // Compute direct radiance
    */

    int numLights = lights.size();
    const int numShadowSamples = ENABLE_PARTICIPATING_MEDIA ? 0 : 4;
    float3 directRadiance = make_float3(0);
    if(numShadowSamples > 0)
    {
        float3 avgLightRadiance = make_float3(0.f);

        for(int shadowSample = 0; shadowSample < numShadowSamples; shadowSample++)
        {
            float sample = getRandomUniformFloat(&randomStates[launchIndex]);
            int randomLightIndex = intmin(int(sample*numLights), lights.size()-1);
            Light & light = lights[randomLightIndex];
            float scale = numLights;
            float3 lightContrib = getLightContribution(light, rec.position, rec.normal, sceneRootObject, randomStates[launchIndex]);
            avgLightRadiance += scale * lightContrib;
        }

        directRadiance = rec.attenuation*avgLightRadiance/numShadowSamples;
    }

    directRadianceBuffer[launchIndex] = directRadiance;
    
}

RT_PROGRAM void gatherAnyHitOnNonEmitter()
{
    shadowPrd.attenuation = 0.0f;
    rtTerminateRay();
}