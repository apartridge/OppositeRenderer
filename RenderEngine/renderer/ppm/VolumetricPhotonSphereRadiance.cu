#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013 Opposite Renderer
 * For the full copyright and license information, please view the LICENSE.txt
 * file that was distributed with this source code.
 */

#include "renderer/ppm/VolumetricRadiancePRD.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>

using namespace optix;

rtDeclareVariable(VolumetricRadiancePRD, volRadiancePrd, rtPayload, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, tHit, rtIntersectionDistance, );

rtDeclareVariable(float3, photonPosition, attribute photonPosition, );
rtDeclareVariable(float3, photonPower, attribute photonPower, );
rtDeclareVariable(uint, photonId, attribute photonId, );

rtDeclareVariable(float, volumetricRadius, , );

RT_PROGRAM void anyHitRadiance()
{
    float t = dot(photonPosition - ray.origin, ray.direction);

    if (t < ray.tmax && t > ray.tmin)
    {
        volRadiancePrd.radiance += (1 / (M_PIf * volumetricRadius * volumetricRadius)) * photonPower
            * exp(-volRadiancePrd.sigma_t * t) * (1.f / (4.f * M_PIf));
        volRadiancePrd.numHits++;
    }
    rtIgnoreIntersection();
}