#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013 Opposite Renderer
 * For the full copyright and license information, please view the LICENSE.txt
 * file that was distributed with this source code.
 */

#include "config.h"
#include "math/Sphere.h"
#include "renderer/Light.h"
#include "renderer/RayType.h"
#include "renderer/ShadowPRD.h"
#include "renderer/helpers/helpers.h"
#include "renderer/helpers/random.h"
#include "renderer/helpers/samplers.h"
#include "renderer/ppm/Photon.h"
#include "renderer/ppm/PhotonPRD.h"
#include <hip/hip_runtime.h>
#include <optix.h>
#include <optixu/optixu_math_namespace.h>

using namespace optix;

rtDeclareVariable(rtObject, sceneRootObject, , );
rtBuffer<Photon, 1> photons;
rtBuffer<RandomState, 2> randomStates;
rtDeclareVariable(uint, maxPhotonDepositsPerEmitted, , );
rtDeclareVariable(uint, photonLaunchWidth, , );
rtBuffer<Light, 1> lights;
rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, );
rtDeclareVariable(uint2, launchDim, rtLaunchDim, );
rtDeclareVariable(Sphere, sceneBoundingSphere, , );

#if ENABLE_RENDER_DEBUG_OUTPUT
rtBuffer<unsigned int, 2> debugPhotonPathLengthBuffer;
#endif

static __device__ void generatePhotonOriginAndDirection(
    const Light& light,
    RandomState& state,
    const Sphere& boundingSphere,
    float3& origin,
    float3& direction,
    float& photonPowerFactor)
{
    origin = light.position;
    float2 sample1 = getRandomUniformFloat2(&state);

    if (light.lightType == Light::AREA)
    {
        float2 sample2 = getRandomUniformFloat2(&state);
        origin += sample1.x * (optix::float3)light.v1 + sample1.y * (optix::float3)light.v2;
        direction = sampleUnitHemisphere(light.normal, sample2);
    }
    else if (light.lightType == Light::POINT)
    {
        // If the point light is well outside the bounding sphere, we make sure to emit
        // only at the scene (to emulate a directional light)
        float3 sceneCenterToLight = light.position - boundingSphere.center;
        float lightDistance = length(sceneCenterToLight);
        sceneCenterToLight /= lightDistance;
        bool lightWellOutsideSphere = (lightDistance > 1.5 * boundingSphere.radius);
        // If light is far away, send photons at the scene and reduce the power based on the solid angle of the scene
        // bounding sphere
        if (lightWellOutsideSphere)
        {
            float3 pointOnDisc = sampleDisc(sample1, boundingSphere.center, boundingSphere.radius, sceneCenterToLight);
            direction = normalize(pointOnDisc - origin);
            // Solid angle of sample disc calculated with http://planetmath.org/calculatingthesolidangleofdisc
            photonPowerFactor
                = (1
                   - lightDistance
                       * rsqrtf(boundingSphere.radius * boundingSphere.radius + lightDistance * lightDistance))
                / 2.f;
        }
        else
        {
            direction = sampleUnitSphere(sample1);
        }
    }
    else if (light.lightType == Light::SPOT)
    {
        float3 pointOnDisc = sampleDisc(sample1, origin + light.direction, sinf(light.angle / 2), light.direction);
        direction = normalize(pointOnDisc - origin);
    }
}

RT_PROGRAM void generator()
{
    PhotonPRD photonPrd;
    photonPrd.pm_index = (launchIndex.y * photonLaunchWidth + launchIndex.x) * maxPhotonDepositsPerEmitted;
    photonPrd.numStoredPhotons = 0;
    photonPrd.depth = 0;
    photonPrd.weight = 1.0f;
    photonPrd.randomState = randomStates[launchIndex];

    int lightIndex = 0;
    if (lights.size() > 1)
    {
        float sample = getRandomUniformFloat(&photonPrd.randomState);
        lightIndex = intmin((int)(sample * lights.size()), lights.size() - 1);
    }

    Light light = lights[lightIndex];
    float powerScale = lights.size();

    photonPrd.power = light.power * powerScale;

    float3 rayOrigin, rayDirection;

    float photonPowerFactor = 1.f;
    generatePhotonOriginAndDirection(
        light, photonPrd.randomState, sceneBoundingSphere, rayOrigin, rayDirection, photonPowerFactor);
    photonPrd.power *= photonPowerFactor;

    Ray photon = Ray(rayOrigin, rayDirection, RayType::PHOTON, 0.0001, RT_DEFAULT_MAX);

#if ACCELERATION_STRUCTURE == ACCELERATION_STRUCTURE_KD_TREE_CPU                                                       \
    || ACCELERATION_STRUCTURE == ACCELERATION_STRUCTURE_UNIFORM_GRID
    // Clear photons owned by this thread
    for (unsigned int i = 0; i < maxPhotonDepositsPerEmitted; ++i)
    {
        photons[photonPrd.pm_index + i].position = make_float3(0.0f);
        photons[photonPrd.pm_index + i].power = make_float3(0.0f);
    }
#endif

    rtTrace(sceneRootObject, photon, photonPrd);

    randomStates[launchIndex] = photonPrd.randomState;

#if ENABLE_RENDER_DEBUG_OUTPUT
    debugPhotonPathLengthBuffer[launchIndex] = photonPrd.depth;
#endif
}

rtDeclareVariable(PhotonPRD, photonPrd, rtPayload, );
RT_PROGRAM void miss()
{
    OPTIX_DEBUG_PRINT(photonPrd.depth, "Photon missed geometry.\n");
}

//
// Exception handler program
//

rtDeclareVariable(float3, exceptionErrorColor, , );
RT_PROGRAM void exception()
{
    // printf("Exception Photon!\n");
    // photonPrd.power = make_float3(0,0,0);
}