#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2013 Opposite Renderer
 * For the full copyright and license information, please view the LICENSE.txt
 * file that was distributed with this source code.
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "config.h"
#include "renderer/Hitpoint.h"
#include "renderer/RayType.h"
#include "renderer/RadiancePRD.h"
#include "renderer/ppm/PhotonPRD.h"
#include "renderer/ppm/Photon.h"
#include "renderer/helpers/random.h"
#include "renderer/helpers/helpers.h"
#include "renderer/helpers/samplers.h"
#include "renderer/helpers/store_photon.h"

using namespace optix;

rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, );
rtDeclareVariable(RadiancePRD, radiancePrd, rtPayload, );
rtDeclareVariable(PhotonPRD, photonPrd, rtPayload, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, tHit, rtIntersectionDistance, );

rtDeclareVariable(float3, geometricNormal, attribute geometricNormal, ); 
rtDeclareVariable(float3, shadingNormal, attribute shadingNormal, ); 

rtBuffer<Photon, 1> photons;
rtBuffer<Hitpoint, 2> raytracePassOutputBuffer;
rtDeclareVariable(rtObject, sceneRootObject, , );
rtDeclareVariable(uint, maxPhotonDepositsPerEmitted, , );
rtDeclareVariable(float3, Kd, , );

#if ACCELERATION_STRUCTURE == ACCELERATION_STRUCTURE_STOCHASTIC_HASH
rtDeclareVariable(uint3, photonsGridSize, , );
rtDeclareVariable(float3, photonsWorldOrigo, ,);
rtDeclareVariable(float, photonsGridCellSize, ,);
rtDeclareVariable(unsigned int, photonsSize,,);
rtBuffer<unsigned int, 1> photonsHashTableCount;
#endif

/*
// Radiance Program
*/

RT_PROGRAM void closestHitRadiance()
{
    float3 worldShadingNormal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shadingNormal ) );
    float3 hitPoint = ray.origin + tHit*ray.direction;

    radiancePrd.flags |= PRD_HIT_NON_SPECULAR;
    radiancePrd.attenuation *= Kd;
    radiancePrd.normal = worldShadingNormal;
    radiancePrd.position = hitPoint;
    radiancePrd.lastTHit = tHit;
    if(radiancePrd.flags & PRD_PATH_TRACING)
    {
        radiancePrd.randomNewDirection = sampleUnitHemisphereCos(worldShadingNormal, getRandomUniformFloat2(&radiancePrd.randomState));
    }
}

/*
// Photon Program
*/

RT_PROGRAM void closestHitPhoton()
{
    float3 worldShadingNormal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shadingNormal ) );
    float3 hitPoint = ray.origin + tHit*ray.direction;
    float3 newPhotonDirection;

    if(photonPrd.depth >= 1 && photonPrd.numStoredPhotons < maxPhotonDepositsPerEmitted)
    {
        Photon photon (photonPrd.power, hitPoint, ray.direction, worldShadingNormal);
        STORE_PHOTON(photon);
    }

    photonPrd.power *= Kd;
    OPTIX_DEBUG_PRINT(photonPrd.depth, "Hit Diffuse P(%.2f %.2f %.2f) RT=%d\n", hitPoint.x, hitPoint.y, hitPoint.z, ray.ray_type);
    photonPrd.weight *= fmaxf(Kd);

    // Use russian roulette sampling from depth X to limit the length of the path

    if( photonPrd.depth >= PHOTON_TRACING_RR_START_DEPTH)
    {
        float probContinue = favgf(Kd);
        float probSample = getRandomUniformFloat(&photonPrd.randomState);
        if(probSample >= probContinue )
        {
            return;
        }
        photonPrd.power /= probContinue;
    }

    photonPrd.depth++;
    if(photonPrd.depth >= MAX_PHOTON_TRACE_DEPTH || photonPrd.weight < 0.001)
    {
        return;
    }

#if ACCELERATION_STRUCTURE == ACCELERATION_STRUCTURE_UNIFORM_GRID || ACCELERATION_STRUCTURE == ACCELERATION_STRUCTURE_KD_TREE_CPU
    if(photonPrd.numStoredPhotons >= maxPhotonDepositsPerEmitted)
        return;
#endif

    newPhotonDirection = sampleUnitHemisphereCos(worldShadingNormal, getRandomUniformFloat2(&photonPrd.randomState));
    optix::Ray newRay( hitPoint, newPhotonDirection, RayType::PHOTON, 0.0001 );
    rtTrace(sceneRootObject, newRay, photonPrd);
}
