#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013 Opposite Renderer
 * For the full copyright and license information, please view the LICENSE.txt
 * file that was distributed with this source code.
 */

#include "renderer/RadiancePRD.h"
#include "renderer/ShadowPRD.h"
#include "renderer/ppm/PhotonPRD.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>

using namespace optix;

rtDeclareVariable(float3, shadingNormal, attribute shadingNormal, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, tHit, rtIntersectionDistance, );
rtDeclareVariable(RadiancePRD, radiancePrd, rtPayload, );
rtDeclareVariable(float3, powerPerArea, , );
rtDeclareVariable(float3, Kd, , );
rtDeclareVariable(ShadowPRD, shadowPrd, rtPayload, );
rtDeclareVariable(PhotonPRD, photonPrd, rtPayload, );

/*
// Radiance Program
*/

RT_PROGRAM void closestHitRadiance()
{
    float3 worldShadingNormal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shadingNormal));
    float3 Le = powerPerArea / M_PIf;
    radiancePrd.radiance += radiancePrd.attenuation * Le;
    radiancePrd.flags |= PRD_HIT_EMITTER;
    radiancePrd.lastTHit = tHit;
}

/*
// Photon Program
*/

RT_PROGRAM void closestHitPhoton()
{
    photonPrd.depth++;
}

RT_PROGRAM void gatherAnyHitOnEmitter()
{
    shadowPrd.attenuation = 1.0f;
    rtTerminateRay();
}