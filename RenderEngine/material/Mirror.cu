#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2013 Opposite Renderer
 * For the full copyright and license information, please view the LICENSE.txt
 * file that was distributed with this source code.
 */

#include "config.h"
#include "renderer/RadiancePRD.h"
#include "renderer/RayType.h"
#include "renderer/ppm/Photon.h"
#include "renderer/ppm/PhotonPRD.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>

using namespace optix;

rtDeclareVariable(uint2, launchIndex, rtLaunchIndex, );
rtDeclareVariable(PhotonPRD, photonPrd, rtPayload, );
rtDeclareVariable(RadiancePRD, radiancePrd, rtPayload, );
rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );
rtDeclareVariable(float, tHit, rtIntersectionDistance, );

rtDeclareVariable(float3, geometricNormal, attribute geometricNormal, );
rtDeclareVariable(float3, shadingNormal, attribute shadingNormal, );

rtDeclareVariable(rtObject, sceneRootObject, , );
rtDeclareVariable(float3, Kr, , );

RT_PROGRAM void closestHitRadiance()
{
    float3 worldShadingNormal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shadingNormal));
    float3 hitPoint = ray.origin + tHit * ray.direction;
    radiancePrd.depth++;
    if (radiancePrd.depth <= MAX_RADIANCE_TRACE_DEPTH)
    {
        radiancePrd.attenuation *= Kr;
        float3 newRayDirection = reflect(ray.direction, worldShadingNormal);
        Ray newRay(hitPoint, newRayDirection, RayType::RADIANCE, 0.0001, RT_DEFAULT_MAX);
        rtTrace(sceneRootObject, newRay, radiancePrd);
    }
    radiancePrd.lastTHit = tHit;
}

RT_PROGRAM void closestHitPhoton()
{
    float3 worldShadingNormal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shadingNormal));
    float3 hitPoint = ray.origin + tHit * ray.direction;
    photonPrd.depth++;
    if (photonPrd.depth <= MAX_PHOTON_TRACE_DEPTH)
    {
        photonPrd.power *= Kr;
        float3 newPhotonDirection = reflect(ray.direction, worldShadingNormal);
        Ray newPhoton(hitPoint, newPhotonDirection, RayType::PHOTON, 0.0001);
        rtTrace(sceneRootObject, newPhoton, photonPrd);
    }
}