#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

 #include <optix_world.h>

 using namespace optix;

 rtDeclareVariable(float4,  sphere, , );

 rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
 rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
 rtDeclareVariable(optix::Ray, ray, rtCurrentRay, );

 template<bool use_robust_method>
 static __device__
 void intersect_sphere(void)
 {
   float3 center = make_float3(sphere);
   float3 O = ray.origin - center;
   float  l = 1 / length(ray.direction);
   float3 D = ray.direction * l;
   float radius = sphere.w;

   float b = dot(O, D);
   float c = dot(O, O)-radius*radius;
   float disc = b*b-c;
   if(disc > 0.0f){
     float sdisc = sqrtf(disc);
     float root1 = (-b - sdisc);

     bool do_refine = false;

     float root11 = 0.0f;

     if(use_robust_method && fabsf(root1) > 10.f * radius) {
       do_refine = true;
     }

     if(do_refine) {
       // refine root1
       float3 O1 = O + root1 * D;
       b = dot(O1, D);
       c = dot(O1, O1) - radius*radius;
       disc = b*b - c;

       if(disc > 0.0f) {
         sdisc = sqrtf(disc);
         root11 = (-b - sdisc);
       }
     }

     bool check_second = true;
     if( rtPotentialIntersection( (root1 + root11) * l ) ) {
       shading_normal = geometric_normal = (O + (root1 + root11)*D)/radius;
       if(rtReportIntersection(0))
         check_second = false;
     }
     if(check_second) {
       float root2 = (-b + sdisc) + (do_refine ? root1 : 0);
       if( rtPotentialIntersection( root2 * l ) ) {
         shading_normal = geometric_normal = (O + root2*D)/radius;
         rtReportIntersection(0);
       }
     }
   }
 }


 RT_PROGRAM void intersect(int primIdx)
 {
   intersect_sphere<false>();
 }


 RT_PROGRAM void robust_intersect(int primIdx)
 {
   intersect_sphere<true>();
 }


 RT_PROGRAM void bounds (int, float result[6])
 {
   const float3 cen = make_float3( sphere );
   const float3 rad = make_float3( sphere.w );

   optix::Aabb* aabb = (optix::Aabb*)result;

   if( rad.x > 0.0f  && !isinf(rad.x) ) {
     aabb->m_min = cen - rad;
     aabb->m_max = cen + rad;
   } else {
     aabb->invalidate();
   }
 }

